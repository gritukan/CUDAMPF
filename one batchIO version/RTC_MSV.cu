#include "hip/hip_runtime.h"
/*
 *
 */

#include "header_def.h"
#include "nvrtc_options.h"		// include <string>

void RTC_MSV(unsigned int number, const char* GPU_kernel, HMMER_PROFILE *hmm,
	     unsigned int *seq_1D, unsigned int *offset, unsigned int *seq_len,
	     unsigned int *iLen, unsigned int sum, double *pVal,
	     int warp, int maxreg, dim3 GRID, dim3 BLOCK)
{	
	/*********************************/
	/* 0. Prepare for cuda drive API */
	/*********************************/
	hipDevice_t cuDevice;
	hipCtx_t context;
	hipModule_t module;
	hipFunction_t kernel;

	checkCudaErrors(hipInit(0));
	checkCudaErrors(hipDeviceGet(&cuDevice, 0));
	checkCudaErrors(hipCtxCreate(&context, 0, cuDevice)); 

	/*********************************************/
	/* 1. Device Property: fixed based on Device */
	/*********************************************/


	/****************************************/
	/* 2. Device Memory Allocation and copy */
	/****************************************/
	StopWatchInterface *timer;
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);

   	/* Driver API pointers */
	hipDeviceptr_t d_seq, d_offset, d_len, d_len_6r, mat_m, score;

	/* Allocation */
	checkCudaErrors(hipMalloc(&d_seq, sum * sizeof(unsigned int)));							/* copy 1D database */
	checkCudaErrors(hipMalloc(&d_offset, number * sizeof(unsigned int)));						/* copy offset of each seq*/
	checkCudaErrors(hipMalloc(&d_len, number * sizeof(unsigned int)));							/* copy raw length of each seq */
	checkCudaErrors(hipMalloc(&d_len_6r, number * sizeof(unsigned int)));						/* copy padding length of each seq */
	checkCudaErrors(hipMalloc(&mat_m, hmm->msvQ * PROTEIN_TYPE * sizeof(__32uint__)));			/* striped EMISSION score */
	checkCudaErrors(hipMalloc(&score, number * sizeof(double)));								/* P-Value as output */

	/* H to D copy */
	checkCudaErrors(hipMemcpyHtoD(d_seq, seq_1D, sum * sizeof(unsigned int)));
	checkCudaErrors(hipMemcpyHtoD(d_offset, offset, number * sizeof(unsigned int)));
	checkCudaErrors(hipMemcpyHtoD(d_len, seq_len, number * sizeof(unsigned int)));
	checkCudaErrors(hipMemcpyHtoD(d_len_6r, iLen, number * sizeof(unsigned int)));
	checkCudaErrors(hipMemcpyHtoD(mat_m, hmm->msv_vec, hmm->msvQ * PROTEIN_TYPE * sizeof(__32uint__)));

	sdkStopTimer(&timer);
    printf("Alloc & H to D Copy time: %f (ms)\n", sdkGetTimerValue(&timer));
    sdkDeleteTimer(&timer);

    /********************************************************/
	/* 3. Runtime compilation, Generate PTX and Load module */
	/********************************************************/
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);

	/* NVRTC create handle */
	hiprtcProgram prog;
	NVRTC_SAFE_CALL("hiprtcCreateProgram", hiprtcCreateProgram(&prog,				// prog
															 GPU_kernel,		// buffer
															 NULL,				// name: CUDA program name. name can be NULL; “default_program” is used when it is NULL.
															 0,					// numHeaders (I put header file path with -I later)
															 NULL,				// headers' content
															 NULL));			// include full name of headers

	/* 1. eliminate const through pointer */
    char *a = NULL;
    const char *b = a;
    const char **opts = &b;

    /* 2. elminate const through reference */
    //char a_value = 'c';
    //char* aa = &a_value;
    //const char *&bb = aa;		// no way with const
    //const char**&ref = aa;	// no way

    /* Dynamic Options */
    char **test_char = new char*[8];

    test_char[0] = new char[__INCLUDE__.length() + strlen("simd_def.h") + 1];					// #include simd_def.h
	strcpy(test_char[0], get_option(__INCLUDE__, "simd_def.h").c_str());

    test_char[1] = new char[__INCLUDE__.length() + strlen("simd_functions.h") + 1];				// #include simd_functions.h
    strcpy(test_char[1], get_option(__INCLUDE__, "simd_functions.h").c_str());

    test_char[2] = new char[__RDC__.length() + __F__.length() + 1];								// -rdc=false
    strcpy(test_char[2], get_option(__RDC__, __F__).c_str());

    test_char[3] = new char[__ARCH__.length() + __CC35__.length() + 1];							// -arch=compute_35
    strcpy(test_char[3], get_option(__ARCH__, __CC35__).c_str());

    test_char[4] = new char[__MAXREG__.length() + int2str(maxreg).length() + 1];				// -maxrregcount = <?>
    strcpy(test_char[4], get_option(__MAXREG__, int2str(maxreg)).c_str());

    test_char[5] = new char[__RIB__.length() + int2str(warp).length() + 1];						// #define RIB <?> : warps per block
    strcpy(test_char[5], get_option(__RIB__, int2str(warp)).c_str());

    test_char[6] = new char[__SIZE__.length() + int2str((int)force_local_size).length() + 1];	// #define SIZE 40
    strcpy(test_char[6], get_option(__SIZE__, int2str((int)force_local_size)).c_str());

    test_char[7] = new char[__Q__.length() + int2str(hmm->msvQ).length() + 1];					// #define Q <?>
    strcpy(test_char[7], get_option(__Q__, int2str(hmm->msvQ)).c_str());

    /* 1. change const char** through pointer */
    //char* **test = const_cast<char** *>(&opts);
    //*test = test_char;

    /* 2. change const char** through reference */
    char** &test_ref = const_cast<char** &>(opts);
    test_ref = test_char;

    /* NVRTC compile */
	NVRTC_SAFE_CALL("hiprtcCompileProgram", hiprtcCompileProgram(prog,	// prog
															   8,		// numOptions
															   opts));	// options

	sdkStopTimer(&timer);
    printf("nvrtc Creat and Compile: %f (ms)\n", sdkGetTimerValue(&timer));
    sdkDeleteTimer(&timer);

	//======================================================================================//
	// /* dump log */																		//	
    // size_t logSize;																		//
    // NVRTC_SAFE_CALL("hiprtcGetProgramLogSize", hiprtcGetProgramLogSize(prog, &logSize));	//
    // char *log = (char *) malloc(sizeof(char) * logSize + 1);								//
    // NVRTC_SAFE_CALL("hiprtcGetProgramLog", hiprtcGetProgramLog(prog, log));				//
    // log[logSize] = '\x0';																//
    // std::cerr << "\n compilation log ---\n";												//
    // std::cerr << log;																	//
    // std::cerr << "\n end log ---\n";														//
    // free(log);																			//
	//======================================================================================//
	
	/* NVRTC fetch PTX */
	sdkCreateTimer(&timer);
    sdkStartTimer(&timer);

	size_t ptxsize;
	NVRTC_SAFE_CALL("hiprtcGetCodeSize", hiprtcGetCodeSize(prog, &ptxsize));
	char *ptx = new char[ptxsize];
	NVRTC_SAFE_CALL("hiprtcGetCode", hiprtcGetCode(prog, ptx));
	NVRTC_SAFE_CALL("hiprtcDestroyProgram", hiprtcDestroyProgram(&prog));	// destroy program instance

	/* Launch PTX by driver API */
	checkCudaErrors(hipModuleLoadDataEx(&module, ptx, 0, 0, 0));
	checkCudaErrors(hipModuleGetFunction(&kernel, module, "KERNEL"));

	sdkStopTimer(&timer);
    printf("Compile & Load time: %f (ms)\n", sdkGetTimerValue(&timer));
    sdkDeleteTimer(&timer);

    /**************************************/
	/* 4. GPU kernel launch by driver API */
	/**************************************/
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);

    /* parameters for kernel funciton */
	void *arr[] = { &d_seq, &number, &d_offset,
					&score, &d_len, &d_len_6r, &mat_m, 
					&(hmm->base_s), &(hmm->bias_s), &(hmm->tbm_s), &(hmm->tec_s),
					&(hmm->scale_b), &(hmm->MU[0]), &(hmm->LAMBDA[0])};

	/* launch kernel */
	checkCudaErrors(hipModuleLaunchKernel(	kernel,
								  	GRID.x, GRID.y, GRID.z,		/* grid dim */
									BLOCK.x, BLOCK.y, BLOCK.z,	/* block dim */
									0,0,						/* SMEM, stream */
									&arr[0],					/* kernel params */
									0));						/* extra opts */

	/* wait for kernel finish */
	checkCudaErrors(hipCtxSynchronize());			/* block for a context's task to complete */

	sdkStopTimer(&timer);
    printf("Kernel time: %f (ms)\n", sdkGetTimerValue(&timer));
    sdkDeleteTimer(&timer);

    /*****************************************/
    /* 5. P-value return and post-processing */
    /*****************************************/
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);

    checkCudaErrors(hipMemcpyDtoH(pVal, score, number * sizeof(double)));

   	sdkStopTimer(&timer);
    printf("D to H copy time: %f (ms)\n", sdkGetTimerValue(&timer));
    sdkDeleteTimer(&timer);

    /* count the number of seqs pass */
	unsigned long n_pass_msv = 0;			/* # of seqs pass MSV filter     */
	unsigned long pass_vit_msv = 0;			/* # of seqs pass msv and vit at same time */

	for (int i = 0; i < number; i++)
	{
		if (pVal[i] <= F2)
			pass_vit_msv++;

		if ((pVal[i] <= F1) && (pVal[i] > F2))
			n_pass_msv++;
	}

	printf("|					PASS MSV					|\n");
	printf("|	 ALL		|	 VIT		|	 FWD		|\n");
	printf("|	%d  	|	%d  	|	%d  	|\n", n_pass_msv + pass_vit_msv, n_pass_msv, pass_vit_msv);

	/************************/
	/* 6. clean the context */
	/************************/
    checkCudaErrors(hipDevicePrimaryCtxReset(cuDevice));		/* reset */
	checkCudaErrors(hipCtxSynchronize());					/* block for a context's task to complete */
}
